#include "hip/hip_runtime.h"
#ifndef SRC_CUDA_UPDATE_ACTIVE_ELEMENT_COUNT_LIST_CU
#define SRC_CUDA_UPDATE_ACTIVE_ELEMENT_COUNT_LIST_CU

__global__ void  UpdateActiveElementOutputCountListKernel( CudaCompactElement*  oldActiveElementList,
                                                           CudaCompactElement*  newActiveElementCountList,
                                                           dim3                 volumeDimensions,
                                                           size_t               oldNumActiveElements );

extern "C" void CudaUpdateActiveElementOutputCountList( CudaCompactElement* oldActiveElementList,
                                                        CudaCompactElement* newActiveElementCountList,
                                                        size_t              oldNumActiveElements,
                                                        dim3                volumeDimensions )
{
    // set the thread block size to the maximum
#ifdef CUDA_ARCH_SM_10   
    dim3 threadBlockDimensions( 128, 1, 1 );
    int numThreadBlocks = static_cast< int >( ceil( oldNumActiveElements / 128.0f ) );
#endif

#ifdef CUDA_ARCH_SM_13
    dim3 threadBlockDimensions( 512, 1, 1 );
    int numThreadBlocks = static_cast< int >( ceil( oldNumActiveElements / 512.0f ) );
#endif

    // set the grid dimensions
    dim3 gridDimensions( numThreadBlocks, 1, 1 );

    if ( numThreadBlocks > 0 )
    {
        // call our kernel
        UpdateActiveElementOutputCountListKernel<<< gridDimensions, threadBlockDimensions >>>(
            oldActiveElementList,
            newActiveElementCountList,
            volumeDimensions,
            oldNumActiveElements );

        CudaSynchronize();
        CudaCheckErrors();
    }
}

__global__ void UpdateActiveElementOutputCountListKernel( CudaCompactElement* oldActiveElementList,
                                                          CudaCompactElement* newActiveElementCountList,
                                                          dim3                volumeDimensions,
                                                          size_t              numActiveVoxels )
{
    int arrayIndexInOldActiveElementList  = ComputeElementCoordinates1D();

    if ( arrayIndexInOldActiveElementList < numActiveVoxels )
    {
        CudaCompactElement packedOldVoxelCoordinate = oldActiveElementList[ arrayIndexInOldActiveElementList ];
        dim3 oldElementCoordinates                  = UnpackCoordinates( packedOldVoxelCoordinate );

        int3 currentCoordinates;
        dim3 currentCoordinatesDim3;
        int  arrayIndex;

        GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u4negativeZ, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 0, -, 1 );
        GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u1,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, -, 1, +, 0 );
        GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u3,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, -, 1, +, 0, +, 0 );
        GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u4,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 0, +, 0 );
        GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u5,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 1, +, 0, +, 0 );
        GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u7,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 1, +, 0 );
        GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u4positiveZ, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 0, +, 1 );

        GET_TAG_NEIGHBORHOOD_HELPER_1D( u4timeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 0, +, 0 );

        bool  outputSelf = false;
        float tolerance  = 0.001f;
        int   numOutputs = 0;

#ifdef TEMPORAL_VOXEL_CULLING
        if ( u4timeDerivative == 1 )
        {
#endif

            if ( !Equals( u4, u1,          tolerance ) ) { outputSelf = true; numOutputs++; }
            if ( !Equals( u4, u3,          tolerance ) ) { outputSelf = true; numOutputs++; }
            if ( !Equals( u4, u5,          tolerance ) ) { outputSelf = true; numOutputs++; }
            if ( !Equals( u4, u7,          tolerance ) ) { outputSelf = true; numOutputs++; }
            if ( !Equals( u4, u4negativeZ, tolerance ) ) { outputSelf = true; numOutputs++; }
            if ( !Equals( u4, u4positiveZ, tolerance ) ) { outputSelf = true; numOutputs++; }

#ifdef TEMPORAL_VOXEL_CULLING
        }
        else
        {
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u1negativeZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, -, 1, -, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u3negativeZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, -, 1, +, 0, -, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u4negativeZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 0, -, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u5negativeZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 1, +, 0, -, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u7negativeZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 1, -, 1 );

            GET_TAG_NEIGHBORHOOD_HELPER_1D( u0timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, -, 1, -, 1, +, 0 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u1timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, -, 1, +, 0 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u2timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 1, -, 1, +, 0 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u3timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, -, 1, +, 0, +, 0 );

            GET_TAG_NEIGHBORHOOD_HELPER_1D( u5timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 1, +, 0, +, 0 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u6timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, -, 1, +, 1, +, 0 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u7timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 1, +, 0 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u8timeDerivative,          currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 1, +, 1, +, 0 );

            GET_TAG_NEIGHBORHOOD_HELPER_1D( u1positiveZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, -, 1, +, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u3positiveZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, -, 1, +, 0, +, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u4positiveZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 0, +, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u5positiveZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 1, +, 0, +, 1 );
            GET_TAG_NEIGHBORHOOD_HELPER_1D( u7positiveZtimeDerivative, currentCoordinates, currentCoordinatesDim3, arrayIndex, oldElementCoordinates, volumeDimensions, +, 0, +, 1, +, 1 );

            //
            // testing -z layer of time derivative neighborhood
            //
            if ( u1negativeZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4negativeZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u1, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }

            if ( u3negativeZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4negativeZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u3, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u4negativeZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4negativeZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u5negativeZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4negativeZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u5, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u7negativeZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4negativeZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u7, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }

            //
            // testing middle layer of time derivative neighborhood
            //
            if ( u0timeDerivative == 1 )
            {
                if ( !Equals( u4, u1, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u3, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u1timeDerivative == 1 )
            {
                if ( !Equals( u4, u1, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u2timeDerivative == 1 )
            {
                if ( !Equals( u4, u1, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u5, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u3timeDerivative == 1 )
            {
                if ( !Equals( u4, u3, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u5timeDerivative == 1 )
            {
                if ( !Equals( u4, u5, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u6timeDerivative == 1 )
            {
                if ( !Equals( u4, u3, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u7, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u7timeDerivative == 1 )
            {
                if ( !Equals( u4, u7, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }

            if ( u8timeDerivative == 1 )
            {
                if ( !Equals( u4, u5, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u7, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            //
            // testing +z layer of time derivative layer
            //
            if ( u1positiveZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4positiveZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u1, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u3positiveZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4positiveZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u3, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u4positiveZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4positiveZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u5positiveZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4positiveZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u5, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }


            if ( u7positiveZtimeDerivative == 1 )
            {
                if ( !Equals( u4, u4positiveZ, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
                if ( !Equals( u4, u7, tolerance ) )
                {
                    outputSelf = true;
                    numOutputs++;
                }
            }
        }
#endif

        if ( outputSelf )
        {
            numOutputs++;
        }

        newActiveElementCountList[ arrayIndexInOldActiveElementList ] = numOutputs;
    }
}


#endif