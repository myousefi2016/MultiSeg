#include "hip/hip_runtime.h"
#ifndef SRC_CUDA_UPDATE_LEVEL_SET_VOLUME_CU
#define SRC_CUDA_UPDATE_LEVEL_SET_VOLUME_CU

template< typename T >
__global__ void  UpdateLevelSetVolumeKernel( CudaLevelSetElement* deviceLevelSetData,
                                             CudaTagElement*      deviceLevelSetExportData,
                                             CudaTagElement*      deviceTimeDerivativeData,
                                             float*               deviceFeatureSpaceDistanceToForegroundVolume,
                                             float*               deviceFeatureSpaceDistanceToBackgroundVolume,                                             
                                             dim3                 volumeDimensions,
                                             size_t               numActiveVoxels,
                                             int                  target,
                                             int                  maxDistanceBeforeShrink,
                                             float                curvatureInfluence,
                                             float                timeStep );

template< typename T >
__device__ void  UpdateLevelSetVolumeKernelHelper( CudaCompactElement   packedVoxelCoordinate,
                                                   CudaLevelSetElement* deviceLevelSetData,
                                                   CudaTagElement*      deviceLevelSetExportData,
                                                   CudaTagElement*      deviceTimeDerivativeData,
                                                   float*               deviceFeatureSpaceDistanceToForegroundVolume,
                                                   float*               deviceFeatureSpaceDistanceToBackgroundVolume,                                                   
                                                   dim3                 volumeDimensions,
                                                   int                  target,
                                                   int                  maxDistanceBeforeShrink,
                                                   float                curvatureInfluence,
                                                   float                timeStep );

extern "C" void CudaUpdateLevelSetVolumeAsync( CudaLevelSetElement*            levelSetData,
                                               CudaTagElement*                 levelSetExportData,
                                               CudaTagElement*                 timeDerivativeData,
                                               thrust::device_vector< float >& featureSpaceDistanceToForegroundVolumeDevice,
                                               thrust::device_vector< float >& featureSpaceDistanceToBackgroundVolumeDevice,                                               
                                               size_t                          numActiveElementsHost,
                                               dim3                            volumeDimensions,
                                               int                             target,
                                               int                             maxDistanceBeforeShrink,
                                               float                           curvatureInfluence,
                                               float                           timeStep,
                                               unsigned int                    numBytesPerVoxel,
                                               bool                            isSigned )
{
#ifdef CUDA_ARCH_SM_10
    // set the thread block size to the maximum
    dim3 threadBlockDimensions( 256, 1, 1 );
    int numThreadBlocks = static_cast< int >( ceil( numActiveElementsHost / ( 256.0f * 4.0f ) ) );
#endif

#ifdef CUDA_ARCH_SM_13
    // set the thread block size to the maximum
    dim3 threadBlockDimensions( 512, 1, 1 );
    int numThreadBlocks = static_cast< int >( ceil( numActiveElementsHost / ( 512.0f * 4.0f ) ) );
#endif

    // set the grid dimensions
    dim3 gridDimensions( numThreadBlocks, 1, 1 );

    if ( numThreadBlocks > 0 )
    {
        // call our kernel
        if ( numBytesPerVoxel == 1 && !isSigned )
        {
            UpdateLevelSetVolumeKernel< unsigned char ><<< gridDimensions, threadBlockDimensions >>>(
                levelSetData,
                levelSetExportData,
                timeDerivativeData,
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToForegroundVolumeDevice[ 0 ] ),
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToBackgroundVolumeDevice[ 0 ] ),
                volumeDimensions,
                numActiveElementsHost,
                target,
                maxDistanceBeforeShrink,
                curvatureInfluence,
                timeStep );
        }
        else
        if ( numBytesPerVoxel == 1 && isSigned )
        {
            UpdateLevelSetVolumeKernel< char ><<< gridDimensions, threadBlockDimensions >>>(
                levelSetData,
                levelSetExportData,
                timeDerivativeData,
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToForegroundVolumeDevice[ 0 ] ),
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToBackgroundVolumeDevice[ 0 ] ),
                volumeDimensions,
                numActiveElementsHost,
                target,
                maxDistanceBeforeShrink,
                curvatureInfluence,
                timeStep );
        }
        else
        if ( numBytesPerVoxel == 2 && !isSigned )
        {
            UpdateLevelSetVolumeKernel< unsigned short ><<< gridDimensions, threadBlockDimensions >>>(
                levelSetData,
                levelSetExportData,
                timeDerivativeData,
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToForegroundVolumeDevice[ 0 ] ),
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToBackgroundVolumeDevice[ 0 ] ),
                volumeDimensions,
                numActiveElementsHost,
                target,
                maxDistanceBeforeShrink,
                curvatureInfluence,
                timeStep );
        }
        else
        if ( numBytesPerVoxel == 2 && isSigned )
        {
            UpdateLevelSetVolumeKernel< short ><<< gridDimensions, threadBlockDimensions >>>(
                levelSetData,
                levelSetExportData,
                timeDerivativeData,
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToForegroundVolumeDevice[ 0 ] ),
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToBackgroundVolumeDevice[ 0 ] ),
                volumeDimensions,
                numActiveElementsHost,
                target,
                maxDistanceBeforeShrink,
                curvatureInfluence,
                timeStep );
        }
        else
        if ( numBytesPerVoxel == 4 && !isSigned )
        {
            UpdateLevelSetVolumeKernel< unsigned int ><<< gridDimensions, threadBlockDimensions >>>(
                levelSetData,
                levelSetExportData,
                timeDerivativeData,
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToForegroundVolumeDevice[ 0 ] ),
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToBackgroundVolumeDevice[ 0 ] ),
                volumeDimensions,
                numActiveElementsHost,
                target,
                maxDistanceBeforeShrink,
                curvatureInfluence,
                timeStep );
        }
        else
        if ( numBytesPerVoxel == 4 && isSigned )
        {
            UpdateLevelSetVolumeKernel< int ><<< gridDimensions, threadBlockDimensions >>>(
                levelSetData,
                levelSetExportData,
                timeDerivativeData,
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToForegroundVolumeDevice[ 0 ] ),
                thrust::raw_pointer_cast< float >( &featureSpaceDistanceToBackgroundVolumeDevice[ 0 ] ),
                volumeDimensions,
                numActiveElementsHost,
                target,
                maxDistanceBeforeShrink,
                curvatureInfluence,
                timeStep );
        }
        else
        {
            Assert( 0 );
        }
    }
}

template< typename T >
__global__ void UpdateLevelSetVolumeKernel( CudaLevelSetElement* deviceLevelSetData,
                                            CudaTagElement*      deviceLevelSetExportData,
                                            CudaTagElement*      deviceTimeDerivativeData,
                                            float*               deviceFeatureSpaceDistanceToForegroundVolume,
                                            float*               deviceFeatureSpaceDistanceToBackgroundVolume,
                                            dim3                 volumeDimensions,
                                            size_t               numActiveVoxels,
                                            int                  target,
                                            int                  maxDistanceBeforeShrink,
                                            float                curvatureInfluence,
                                            float                timeStep )
{
    int arrayIndexInActiveElementTexture = ComputeIndexThread1DBlock1DTo1D();
    int arrayIndexInActiveElementList    = arrayIndexInActiveElementTexture * 4;

    if ( arrayIndexInActiveElementList < numActiveVoxels )
    {
        CudaCompactElement4 packedCoordinates = tex1Dfetch( CUDA_TEXTURE_REF_ACTIVE_ELEMENTS_1D, arrayIndexInActiveElementTexture );

        UpdateLevelSetVolumeKernelHelper< T >( packedCoordinates.x,
                                               deviceLevelSetData,
                                               deviceLevelSetExportData,
                                               deviceTimeDerivativeData,
                                               deviceFeatureSpaceDistanceToForegroundVolume,
                                               deviceFeatureSpaceDistanceToBackgroundVolume,
                                               volumeDimensions,
                                               target,
                                               maxDistanceBeforeShrink,
                                               curvatureInfluence,
                                               timeStep );                
        arrayIndexInActiveElementList++;

        if ( arrayIndexInActiveElementList < numActiveVoxels )
        {
            UpdateLevelSetVolumeKernelHelper< T >( packedCoordinates.y,
                                                   deviceLevelSetData,
                                                   deviceLevelSetExportData,
                                                   deviceTimeDerivativeData,
                                                   deviceFeatureSpaceDistanceToForegroundVolume,
                                                   deviceFeatureSpaceDistanceToBackgroundVolume,
                                                   volumeDimensions,
                                                   target,
                                                   maxDistanceBeforeShrink,
                                                   curvatureInfluence,
                                                   timeStep );                
            arrayIndexInActiveElementList++;

            if ( arrayIndexInActiveElementList < numActiveVoxels )
            {
                UpdateLevelSetVolumeKernelHelper< T >( packedCoordinates.z,
                                                       deviceLevelSetData,
                                                       deviceLevelSetExportData,
                                                       deviceTimeDerivativeData,
                                                       deviceFeatureSpaceDistanceToForegroundVolume,
                                                       deviceFeatureSpaceDistanceToBackgroundVolume,
                                                       volumeDimensions,
                                                       target,
                                                       maxDistanceBeforeShrink,
                                                       curvatureInfluence,
                                                       timeStep );                
                arrayIndexInActiveElementList++;

                if ( arrayIndexInActiveElementList < numActiveVoxels )
                {
                    UpdateLevelSetVolumeKernelHelper< T >( packedCoordinates.w,
                                                           deviceLevelSetData,
                                                           deviceLevelSetExportData,
                                                           deviceTimeDerivativeData,
                                                           deviceFeatureSpaceDistanceToForegroundVolume,
                                                           deviceFeatureSpaceDistanceToBackgroundVolume,
                                                           volumeDimensions,
                                                           target,
                                                           maxDistanceBeforeShrink,
                                                           curvatureInfluence,
                                                           timeStep );                
                }
            }
        }
    }
}

__device__ float ComputeDensityTerm( float        featureSpaceDistanceToForeground,
                                     unsigned int maxDifferenceBeforeShrink )
{
    float f = ( -1.0f / maxDifferenceBeforeShrink ) * abs( featureSpaceDistanceToForeground ) + 1.0f;

    return f;
}

template< typename T >
__device__ void  UpdateLevelSetVolumeKernelHelper( CudaCompactElement   packedVoxelCoordinate,
                                                   CudaLevelSetElement* deviceLevelSetData,
                                                   CudaTagElement*      deviceLevelSetExportData,
                                                   CudaTagElement*      deviceTimeDerivativeData,
                                                   float*               deviceFeatureSpaceDistanceToForegroundVolume,
                                                   float*               deviceFeatureSpaceDistanceToBackgroundVolume,
                                                   dim3                 volumeDimensions,
                                                   int                  target,
                                                   int                  maxDistanceBeforeShrink,
                                                   float                curvatureInfluence,
                                                   float                timeStep )
{

    dim3 elementCoordinates = UnpackCoordinates( packedVoxelCoordinate );

    //
    // get 3x3x3 neighborhood
    //
    int3 currentCoordinates;
    dim3 currentCoordinatesDim3;
    int  arrayIndex;
    int  elementIndex;
    int  elementIndexUntiled;

    elementIndex        = ComputeIndex3DToTiled1D( elementCoordinates, volumeDimensions );
    elementIndexUntiled = ComputeIndex3DTo1D( elementCoordinates, volumeDimensions );

    GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u4negativeZ, currentCoordinatesDim3, arrayIndex, elementCoordinates, volumeDimensions, +, 0, +, 0, -, 1 );
                                                                         
    GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u1,          currentCoordinatesDim3, arrayIndex, elementCoordinates, volumeDimensions, +, 0, -, 1, +, 0 );
    GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u3,          currentCoordinatesDim3, arrayIndex, elementCoordinates, volumeDimensions, -, 1, +, 0, +, 0 );
    GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u4,          currentCoordinatesDim3, arrayIndex, elementCoordinates, volumeDimensions, +, 0, +, 0, +, 0 );
    GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u5,          currentCoordinatesDim3, arrayIndex, elementCoordinates, volumeDimensions, +, 1, +, 0, +, 0 );
    GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u7,          currentCoordinatesDim3, arrayIndex, elementCoordinates, volumeDimensions, +, 0, +, 1, +, 0 );
                                                                         
    GET_LEVEL_SET_NEIGHBORHOOD_HELPER_1D( u4positiveZ, currentCoordinatesDim3, arrayIndex, elementCoordinates, volumeDimensions, +, 0, +, 0, +, 1 );

    GET_LEFT_SOURCE_NEIGHBORHOOD_HELPER ( u4LeftSource,  currentCoordinates, elementCoordinates, volumeDimensions, +, 0, +, 0, +, 0, T );
    GET_RIGHT_SOURCE_NEIGHBORHOOD_HELPER( u4RightSource, currentCoordinates, elementCoordinates, volumeDimensions, +, 0, +, 0, +, 0, T );

    float featureSpaceDistanceToForeground = deviceFeatureSpaceDistanceToForegroundVolume[ elementIndexUntiled ];
    float featureSpaceDistanceToBackground = deviceFeatureSpaceDistanceToForegroundVolume[ elementIndexUntiled ];
    
    // compute derivatives
    float derivativeCentralX          = ( u5 - u3 )                   / 2.0f;
    float derivativeCentralY          = ( u7 - u1 )                   / 2.0f;
    float derivativeCentralZ          = ( u4positiveZ - u4negativeZ ) / 2.0f; 

    float derivativeForwardX          = ( u5          - u4 );
    float derivativeForwardY          = ( u7          - u4 );
    float derivativeForwardZ          = ( u4positiveZ - u4 );

    float derivativeBackwardX         = ( u4 - u3 );
    float derivativeBackwardY         = ( u4 - u1 );
    float derivativeBackwardZ         = ( u4 - u4negativeZ );

    float secondDerivativeX = derivativeForwardX - derivativeBackwardX;
    float secondDerivativeY = derivativeForwardY - derivativeBackwardY;
    float secondDerivativeZ = derivativeForwardZ - derivativeBackwardZ;

    // compute speed function terms
    float connectivityTerm = 0.0f;

    if ( curvatureInfluence > 0.0f )
    {
        if ( Equals( u4positiveZ, u4negativeZ, 0.1f ) &&            
             Equals( u4negativeZ, u1,          0.1f ) &&
             Equals( u1, u3,                   0.1f ) &&
             Equals( u3, u5,                   0.1f ) &&
             Equals( u5, u7,                   0.1f ) &&
             Equals( u7, u4positiveZ,          0.1f ) )
        {
            float average    = ( u4positiveZ + u1 + u3 + u5 + u7 + u4negativeZ ) / 6.0f;
            connectivityTerm = average - u4;
        }
    }

    float curvatureTerm  = ( secondDerivativeX + secondDerivativeY + secondDerivativeZ );
    
    float densityTerm    = - ComputeDensityTerm( featureSpaceDistanceToForeground, maxDistanceBeforeShrink );
    
    if ( timeStep < 0 )
    {
        timeStep    = - timeStep;
        densityTerm = - densityTerm;
    }

    float speedFunction  = ( ( 1 - curvatureInfluence ) * ( densityTerm ) ) + ( ( curvatureInfluence ) * curvatureTerm );

    // gradient magnitude
    float gradientLength = sqrt( Sqr( derivativeCentralX ) + Sqr( derivativeCentralY ) + Sqr( derivativeCentralZ ) );

    // compute new value
    float levelSetDelta    = ( ( timeStep * speedFunction * gradientLength ) ) + connectivityTerm;
    float newLevelSetValue = min( 1.0f, max( -1.0f, u4 + levelSetDelta ) );

    // rescale
    newLevelSetValue = min( 1.0f, max( -1.0f, ( ( ( 1.0f + LEVEL_SET_RESCALE_AMOUNT ) * newLevelSetValue ) + LEVEL_SET_RESCALE_AMOUNT ) ) );

    // assign new value
#ifdef LEVEL_SET_FIELD_FIXED_POINT
    int writeValue                     = __float2int_rd( newLevelSetValue * LEVEL_SET_FIELD_MAX_VALUE );
    deviceLevelSetData[ elementIndex ] = writeValue;
#else
    deviceLevelSetData[ elementIndex ] = newLevelSetValue;
#endif

    int exportValue                                 = __float2int_rd( newLevelSetValue * LEVEL_SET_FIELD_EXPORT_MAX_VALUE );
    deviceLevelSetExportData[ elementIndexUntiled ] = exportValue;


    // compute and assign time derivative
    float timeDerivative = newLevelSetValue - u4;

    int timeDerivativeValue = abs( timeDerivative ) > TEMPORAL_DERIVATIVE_THRESHOLD ? 1 : 0;
    deviceTimeDerivativeData[ elementIndex ] = timeDerivativeValue;
}






struct Index1DToIndex3DFunction : public thrust::unary_function< int, int3 >
{
    int mNumVoxelsX, mNumVoxelsXY;
    float mInvNumVoxelsX, mInvNumVoxelsXY;

    Index1DToIndex3DFunction( int3 numVoxels ) :
        mNumVoxelsX    ( numVoxels.x ),
        mNumVoxelsXY   ( numVoxels.x * numVoxels.y ),
        mInvNumVoxelsX ( 1.0f / (float)mNumVoxelsX ),
        mInvNumVoxelsXY( 1.0f / (float)mNumVoxelsXY )
    {}

    __device__
    int3 operator() ( int index1D )
    {
        int z        = __float2uint_rd( index1D * mInvNumVoxelsXY );
        index1D      = index1D - ( z * mNumVoxelsXY );
        int y        = __float2uint_rd( index1D * mInvNumVoxelsX );
        int x        = index1D - ( y * mNumVoxelsX );

        int3 index3D = make_int3( x, y, z );

        return index3D;
    }
};

__constant__ CudaConstraintValues constantMemoryCudaConstraintValues;

struct InitializeFeatureSpaceDistanceMapsFunction
{
    enum TupleLayout
    {
        INDEX_3D,
        FEATURE_SPACE_DISTANCE_TO_FOREGROUND,
        FEATURE_SPACE_DISTANCE_TO_BACKGROUND        
    };

    InitializeFeatureSpaceDistanceMapsFunction() {}

    template < typename TTuple >
    __device__ void operator() ( TTuple tuple )
    {
        int3  index3D            = thrust::get< INDEX_3D >( tuple );
        int3  elementCoordinates = index3D;
        int3  currentCoordinates;
        
        GET_LEFT_SOURCE_NEIGHBORHOOD_HELPER ( u4LeftSource,  currentCoordinates, elementCoordinates, volumeDimensions, +, 0, +, 0, +, 0, unsigned char );
        GET_RIGHT_SOURCE_NEIGHBORHOOD_HELPER( u4RightSource, currentCoordinates, elementCoordinates, volumeDimensions, +, 0, +, 0, +, 0, unsigned char );

        float2 referenceFeatureSpacePoint = make_float2( u4LeftSource, u4RightSource );

        float minFeatureSpaceDistanceToForeground = 999999.0f;
        float minFeatureSpaceDistanceToBackground = 999999.0f;
        
        //
        // foreground
        //
        for ( int i = 0; i < constantMemoryCudaConstraintValues.numForegroundValues; i++ )
        {
            float2 currentFeatureSpaceForegroundPoint      = make_float2( constantMemoryCudaConstraintValues.leftForegroundValues[ i ], constantMemoryCudaConstraintValues.rightForegroundValues[ i ] );
            float  currentFeatureSpaceDistanceToForeground = length( referenceFeatureSpacePoint - currentFeatureSpaceForegroundPoint );
            
            if ( currentFeatureSpaceDistanceToForeground < minFeatureSpaceDistanceToForeground )
            {
                minFeatureSpaceDistanceToForeground = currentFeatureSpaceDistanceToForeground;
            }
        }
        
        //
        // background
        //
       for ( int i = 0; i < constantMemoryCudaConstraintValues.numBackgroundValues; i++ )
        {
            float2 currentFeatureSpaceBackgroundPoint      = make_float2( constantMemoryCudaConstraintValues.leftBackgroundValues[ i ], constantMemoryCudaConstraintValues.rightBackgroundValues[ i ] );
            float  currentFeatureSpaceDistanceToBackground = length( referenceFeatureSpacePoint - currentFeatureSpaceBackgroundPoint );
            
            if ( currentFeatureSpaceDistanceToBackground < minFeatureSpaceDistanceToBackground )
            {
                minFeatureSpaceDistanceToBackground = currentFeatureSpaceDistanceToBackground;
            }
        }
                
        thrust::get< FEATURE_SPACE_DISTANCE_TO_FOREGROUND >( tuple ) = minFeatureSpaceDistanceToForeground;
        thrust::get< FEATURE_SPACE_DISTANCE_TO_BACKGROUND >( tuple ) = minFeatureSpaceDistanceToBackground;
    }
};

extern "C" void CudaInitializeFeatureSpaceDistanceMaps( CudaConstraintValues*           cudaConstraintValues,
                                                        thrust::device_vector< float >& featureSpaceDistanceToForegroundVolumeDevice,
                                                        thrust::device_vector< float >& featureSpaceDistanceToBackgroundVolumeDevice, 
                                                        dim3                            volumeDimensions )
{
    int  numElements          = volumeDimensions.x * volumeDimensions.y * volumeDimensions.z;
    int3 volumeDimensionsInt3 = make_int3( volumeDimensions.x, volumeDimensions.y, volumeDimensions.z );
    
    MOJO_CUDA_SAFE( hipMemcpyToSymbol(HIP_SYMBOL( constantMemoryCudaConstraintValues), cudaConstraintValues, sizeof( CudaConstraintValues ) ) );
    
    MOJO_THRUST_SAFE(
        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple( 
                    thrust::make_transform_iterator(
                        thrust::make_counting_iterator( 0 ),
                        Index1DToIndex3DFunction( volumeDimensionsInt3 ) ),
                    featureSpaceDistanceToForegroundVolumeDevice.begin(),
                    featureSpaceDistanceToBackgroundVolumeDevice.begin() ) ),

            thrust::make_zip_iterator(
                thrust::make_tuple( 
                    thrust::make_transform_iterator( 
                        thrust::make_counting_iterator( numElements ),
                        Index1DToIndex3DFunction( volumeDimensionsInt3 ) ),
                    featureSpaceDistanceToForegroundVolumeDevice.end(),
                    featureSpaceDistanceToBackgroundVolumeDevice.end() ) ),
              
            InitializeFeatureSpaceDistanceMapsFunction() ) );
}
                                                        
#endif
 